#include <hip/hip_runtime.h>
#include <stdint.h>
#include <string.h>


namespace sha256 {
    // ... (保持原有SHA256实现不变)
    __device__ const uint32_t k[] = {
        0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
        0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
        0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
        0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
        0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
        0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
        0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
        0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
    };


    __device__ uint32_t rotr(uint32_t x, int n) {
        return (x >> n) | (x << (32 - n));
    }


    __device__ void transform(uint32_t state[8], const uint8_t block[64]) {
        uint32_t w[64];
        uint32_t a, b, c, d, e, f, g, h, t1, t2;


        for (int i = 0; i < 16; i++) {
            w[i] = (block[4*i] << 24) | (block[4*i+1] << 16) | (block[4*i+2] << 8) | block[4*i+3];
        }
        for (int i = 16; i < 64; i++) {
            uint32_t s0 = rotr(w[i-15], 7) ^ rotr(w[i-15], 18) ^ (w[i-15] >> 3);
            uint32_t s1 = rotr(w[i-2], 17) ^ rotr(w[i-2], 19) ^ (w[i-2] >> 10);
            w[i] = w[i-16] + s0 + w[i-7] + s1;
        }

        a = state[0]; b = state[1]; c = state[2]; d = state[3];
        e = state[4]; f = state[5]; g = state[6]; h = state[7];

        for (int i = 0; i < 64; i++) {
            uint32_t S1 = rotr(e, 14) ^ rotr(e, 18) ^ rotr(e, 41);
            uint32_t ch = (e & f) ^ (~e & g);
            t1 = h + S1 + ch + k[i] + w[i];
            uint32_t S0 = rotr(a, 2) ^ rotr(a, 13) ^ rotr(a, 22);
            uint32_t maj = (a & b) ^ (a & c) ^ (b & c);
            t2 = S0 + maj;

            h = g; g = f; f = e; e = d + t1;
            d = c; c = b; b = a; a = t1 + t2;
        }


        state[0] += a; state[1] += b; state[2] += c; state[3] += d;
        state[4] += e; state[5] += f; state[6] += g; state[7] += h;
    }


    __device__ void sha256(uint8_t *out, const uint8_t *in, size_t len) {
        uint32_t state[8] = {
            0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
            0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
        };
        uint8_t block[64];
        size_t off = 0;


        while (off + 64 <= len) {
            memcpy(block, in + off, 64);
            transform(state, block);
            off += 64;
        }


        memcpy(block, in + off, len - off);
        size_t rem = len - off;
        block[rem++] = 0x80;


        if (rem <= 56) {
            memset(block + rem, 0, 56 - rem);
        } else {
            memset(block + rem, 0, 64 - rem);
            transform(state, block);
            memset(block, 0, 56);
        }


        uint64_t bits = len * 8;
        block[56] = (bits >> 56) & 0xff;
        block[57] = (bits >> 48) & 0xff;
        block[58] = (bits >> 40) & 0xff;
        block[59] = (bits >> 32) & 0xff;
        block[60] = (bits >> 24) & 0xff;
        block[61] = (bits >> 16) & 0xff;
        block[62] = (bits >> 8) & 0xff;
        block[63] = bits & 0xff;
        transform(state, block);


        for (int i = 0; i < 8; i++) {
            out[4*i] = (state[i] >> 24) & 0xff;
            out[4*i+1] = (state[i] >> 16) & 0xff;
            out[4*i+2] = (state[i] >> 8) & 0xff;
            out[4*i+3] = state[i] & 0xff;
        }
    }


    __device__ void sha256d(uint8_t *out, const uint8_t *in, size_t len) {
        uint8_t buf[32];
        sha256(buf, in, len);
        sha256(out, buf, 32);
    }
}

#pragma pack(push, 1)
struct BlockHeader {
    uint32_t version;
    uint8_t previousHash[32];
    uint8_t merkleRoot[32];
    uint32_t time;
    uint32_t difficultyTarget;
    uint32_t nonce;
};
#pragma pack(pop)


__device__ void bitsToTarget(uint32_t bits, uint8_t* target) {
    memset(target, 0, 32);
    uint8_t exponent = (bits >> 24) & 0xFF;
    uint32_t mantissa = bits & 0x00FFFFFF;

    if (exponent < 3) {
        mantissa >>= 8 * (3 - exponent);
        exponent = 3;
    }
    if (exponent > 32) exponent = 32;

    int shift = (exponent - 3) * 8;
    for (int i = 0; i < 3; i++) {
        int pos = shift / 8 + (2 - i);
        if (pos < 32) {
            target[pos] = (mantissa >> (i * 8)) & 0xFF;
        }
    }
}


__device__ bool isValidHash(uint8_t* hash, uint32_t bits) {
    uint8_t target[32];
    bitsToTarget(bits, target);

    for (int i = 0; i < 32; i++) {
        if (hash[i] < target[i]) return true;
        if (hash[i] > target[i]) return false;
    }
    return true;
}

// 调试辅助函数：将字节数组转换为十六进制字符串（设备端）
__device__ void bytesToHex(const uint8_t* bytes, size_t len, char* hex) {
    const char* hexChars = "0123456789abcdef";
    for (size_t i = 0; i < len; i++) {
        hex[i*2] = hexChars[(bytes[i] >> 4) & 0x0F];
        hex[i*2+1] = hexChars[bytes[i] & 0x0F];
    }
    hex[len*2] = '\0';
}

// 序列化BlockHeader到字节数组（关键修正：哈希字节序处理）
__device__ void serializeHeader(BlockHeader* header, uint8_t* buffer) {
    // 1. 版本号（4字节，小端）
    buffer[0] = (header->version) & 0xFF;
    buffer[1] = (header->version >> 8) & 0xFF;
    buffer[2] = (header->version >> 16) & 0xFF;
    buffer[3] = (header->version >> 24) & 0xFF;

    // 2. 前区块哈希（32字节，关键修正：不反转哈希字节序）
    // 原因：Java端接收的已经是反转后的哈希，直接复制即可
    memcpy(buffer + 4, header->previousHash, 32);

    // 3. 默克尔根（32字节，关键修正：不反转哈希字节序）
    memcpy(buffer + 36, header->merkleRoot, 32);

    // 4. 时间戳（4字节，小端）
    buffer[68] = (header->time) & 0xFF;
    buffer[69] = (header->time >> 8) & 0xFF;
    buffer[70] = (header->time >> 16) & 0xFF;
    buffer[71] = (header->time >> 24) & 0xFF;

    // 5. 难度目标（4字节，小端）
    buffer[72] = (header->difficultyTarget) & 0xFF;
    buffer[73] = (header->difficultyTarget >> 8) & 0xFF;
    buffer[74] = (header->difficultyTarget >> 16) & 0xFF;
    buffer[75] = (header->difficultyTarget >> 24) & 0xFF;

    // 6. Nonce（4字节，小端）
    buffer[76] = (header->nonce) & 0xFF;
    buffer[77] = (header->nonce >> 8) & 0xFF;
    buffer[78] = (header->nonce >> 16) & 0xFF;
    buffer[79] = (header->nonce >> 24) & 0xFF;
}


__device__ uint8_t debugHeader[80];
__device__ char debugHeaderHex[161];  // 80字节 * 2 + 1
__device__ uint32_t foundNonce = 0;
__device__ int found = 0;
__device__ uint8_t foundHash[32];
__device__ char foundHashHex[65];     // 32字节 * 2 + 1

extern "C" __global__ void miningKernel(
    BlockHeader* header,
    uint32_t startNonce,
    uint32_t endNonce
) {
    uint32_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t totalThreads = gridDim.x * blockDim.x;
    uint32_t currentNonce = startNonce + threadId;

    BlockHeader localHeader = *header;

    // 线程0负责生成调试信息（用于对比Java序列化结果）
    if (threadId == 0) {
        serializeHeader(&localHeader, debugHeader);
    }

    uint8_t headerBuffer[80];
    uint8_t hash[32];

    while (currentNonce < endNonce) {
        if (found) return;

        localHeader.nonce = currentNonce;
        serializeHeader(&localHeader, headerBuffer);
        sha256::sha256d(hash, headerBuffer, 80);

        if (isValidHash(hash, localHeader.difficultyTarget)) {
            int expected = 0;
            if (atomicCAS(&found, expected, 1) == expected) {
                foundNonce = currentNonce;
                memcpy(foundHash, hash, 32);
                return;
            }
        }

        currentNonce += totalThreads;
    }
}
