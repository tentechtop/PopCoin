#include <hip/hip_runtime.h>
#include <stdint.h>
#include <string.h>

// SHA-256d实现（双SHA-256，适配GPU）
namespace sha256 {
    // 常量定义（SHA-256标准）
    __device__ const uint32_t k[] = {
        0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
        0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
        0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
        0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
        0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
        0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
        0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
        0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
    };

    // 旋转右移
    __device__ uint32_t rotr(uint32_t x, int n) {
        return (x >> n) | (x << (32 - n));
    }

    // 加密过程
    __device__ void transform(uint32_t state[8], const uint8_t block[64]) {
        uint32_t w[64];
        uint32_t a, b, c, d, e, f, g, h, t1, t2;

        // 初始化消息调度数组
        for (int i = 0; i < 16; i++) {
            w[i] = (block[4*i] << 24) | (block[4*i+1] << 16) | (block[4*i+2] << 8) | block[4*i+3];
        }
        for (int i = 16; i < 64; i++) {
            uint32_t s0 = rotr(w[i-15], 7) ^ rotr(w[i-15], 18) ^ (w[i-15] >> 3);
            uint32_t s1 = rotr(w[i-2], 17) ^ rotr(w[i-2], 19) ^ (w[i-2] >> 10);
            w[i] = w[i-16] + s0 + w[i-7] + s1;
        }

        // 初始化工作变量
        a = state[0]; b = state[1]; c = state[2]; d = state[3];
        e = state[4]; f = state[5]; g = state[6]; h = state[7];

        // 主循环
        for (int i = 0; i < 64; i++) {
            uint32_t S1 = rotr(e, 14) ^ rotr(e, 18) ^ rotr(e, 41);
            uint32_t ch = (e & f) ^ (~e & g);
            t1 = h + S1 + ch + k[i] + w[i];
            uint32_t S0 = rotr(a, 2) ^ rotr(a, 13) ^ rotr(a, 22);
            uint32_t maj = (a & b) ^ (a & c) ^ (b & c);
            t2 = S0 + maj;

            h = g; g = f; f = e; e = d + t1;
            d = c; c = b; b = a; a = t1 + t2;
        }

        // 更新状态
        state[0] += a; state[1] += b; state[2] += c; state[3] += d;
        state[4] += e; state[5] += f; state[6] += g; state[7] += h;
    }

    // 计算单轮SHA-256
    __device__ void sha256(uint8_t *out, const uint8_t *in, size_t len) {
        uint32_t state[8] = {
            0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
            0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
        };
        uint8_t block[64];
        size_t off = 0;

        // 处理完整的512位块
        while (off + 64 <= len) {
            memcpy(block, in + off, 64);
            transform(state, block);
            off += 64;
        }

        // 处理剩余数据
        memcpy(block, in + off, len - off);
        size_t rem = len - off;
        block[rem++] = 0x80;

        // 填充
        if (rem <= 56) {
            memset(block + rem, 0, 56 - rem);
        } else {
            memset(block + rem, 0, 64 - rem);
            transform(state, block);
            memset(block, 0, 56);
        }

        // 附加长度（bits）
        uint64_t bits = len * 8;
        block[56] = (bits >> 56) & 0xff;
        block[57] = (bits >> 48) & 0xff;
        block[58] = (bits >> 40) & 0xff;
        block[59] = (bits >> 32) & 0xff;
        block[60] = (bits >> 24) & 0xff;
        block[61] = (bits >> 16) & 0xff;
        block[62] = (bits >> 8) & 0xff;
        block[63] = bits & 0xff;
        transform(state, block);

        // 输出结果（大端转小端）
        for (int i = 0; i < 8; i++) {
            out[4*i] = (state[i] >> 24) & 0xff;
            out[4*i+1] = (state[i] >> 16) & 0xff;
            out[4*i+2] = (state[i] >> 8) & 0xff;
            out[4*i+3] = state[i] & 0xff;
        }
    }

    // 双SHA-256（比特币标准）
    __device__ void sha256d(uint8_t *out, const uint8_t *in, size_t len) {
        uint8_t buf[32];
        sha256(buf, in, len);  // 第一次哈希
        sha256(out, buf, 32);  // 第二次哈希
    }
}

// 区块头数据结构（与Java序列化结果严格匹配，80字节）
struct BlockHeader {
    uint32_t version;          // 4字节，小端（Java: Integer.reverseBytes()）
    uint8_t previousHash[32];  // 32字节，小端（Java: reverseBytes()处理后）
    uint8_t merkleRoot[32];    // 32字节，小端（Java: reverseBytes()处理后）
    uint32_t time;             // 4字节，小端（Java: 秒级时间戳，Integer.reverseBytes()）
    uint32_t difficultyTarget; // 4字节，小端（压缩难度，Java: reverseBytes()处理后）
    uint32_t nonce;            // 4字节，小端（Java: Integer.reverseBytes()）
};

// 全局变量：挖矿结果（GPU全局内存）
__device__ uint32_t foundNonce = 0;
__device__ int found = 0;  // 0=未找到，1=已找到
__device__ uint8_t foundHash[32];

// 辅助函数：将压缩难度（bits）转换为256位目标值（小端存储）
__device__ void bitsToTarget(uint32_t bits, uint8_t* target) {
    memset(target, 0, 32);
    uint8_t exponent = (bits >> 24) & 0xFF;  // 指数部分
    uint32_t mantissa = bits & 0x00FFFFFF;   // 尾数部分（大端）

    // 处理指数范围
    if (exponent < 3) {
        mantissa >>= 8 * (3 - exponent);
        exponent = 3;
    }
    if (exponent > 32) exponent = 32;

    // 转换为小端存储的256位目标值
    int shift = (exponent - 3) * 8;  // 位移量（字节）
    for (int i = 0; i < 3; i++) {
        int pos = shift / 8 + (2 - i);  // 小端存储位置
        if (pos < 32) {
            target[pos] = (mantissa >> (i * 8)) & 0xFF;
        }
    }
}

// 辅助函数：验证哈希是否满足难度目标（均为小端比较）
__device__ bool isValidHash(uint8_t* hash, uint32_t bits) {
    uint8_t target[32];
    bitsToTarget(bits, target);

    // 逐字节比较（小端存储，哈希 <= 目标即为有效）
    for (int i = 0; i < 32; i++) {
        if (hash[i] < target[i]) return true;
        if (hash[i] > target[i]) return false;
    }
    return true;  // 等于目标值时有效
}

// GPU挖矿内核（优化RTX 4060并行效率）
__global__ void mineKernel(BlockHeader header, uint32_t startNonce, uint32_t endNonce) {
    // 共享内存缓存（减少全局内存访问延迟）
    __shared__ uint8_t s_prevHash[32];
    __shared__ uint8_t s_merkleRoot[32];
    __shared__ uint32_t s_version;
    __shared__ uint32_t s_time;
    __shared__ uint32_t s_bits;

    // 线程0负责初始化共享内存
    if (threadIdx.x == 0) {
        memcpy(s_prevHash, header.previousHash, 32);
        memcpy(s_merkleRoot, header.merkleRoot, 32);
        s_version = header.version;
        s_time = header.time;
        s_bits = header.difficultyTarget;
    }
    __syncthreads();  // 等待共享内存初始化完成

    // 计算线程唯一ID和对应的nonce
    uint32_t globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t nonce = startNonce + globalThreadId;

    // 已找到结果或超出范围则退出
    if (found || nonce >= endNonce) return;

    // 构造本地区块头（使用共享内存数据）
    BlockHeader localHeader;
    localHeader.version = s_version;
    memcpy(localHeader.previousHash, s_prevHash, 32);
    memcpy(localHeader.merkleRoot, s_merkleRoot, 32);
    localHeader.time = s_time;
    localHeader.difficultyTarget = s_bits;
    localHeader.nonce = nonce;

    // 计算双SHA-256哈希
    uint8_t hash[32];
    sha256::sha256d(hash, (uint8_t*)&localHeader, sizeof(BlockHeader));

    // 验证哈希是否有效
    if (isValidHash(hash, s_bits)) {
        // 原子操作确保只有第一个找到的结果被记录
        if (atomicExch(&found, 1) == 0) {
            foundNonce = nonce;
            memcpy(foundHash, hash, 32);
        }
    }
}
